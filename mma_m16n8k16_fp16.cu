#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <random>

#include "cuda_utils.cuh"

void GenerateRandomData(half *matrix, int m, int n) {
    assert(matrix != nullptr);
    assert(m >= 0);
    assert(n >= 0);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            matrix[i * n + j] = (half)dis(gen);
        }
    }
}

void Transpose2D(half *matrix, const int M, const int N) {
    assert(matrix != nullptr);
    auto buffer = new half[M * N]();
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            buffer[n * M + m] = matrix[m * N + n];
        }
    }
    memcpy(matrix, buffer, M * N * sizeof(half));
    delete[] buffer;
}

void Gemm(const half *matrix_a, const half *matrix_b, half *matrix_c, const int M, const int N, const int K) {
    assert(matrix_a != nullptr);
    assert(matrix_b != nullptr);
    assert(matrix_c != nullptr);
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float res = 0.0f;
            for (int k = 0; k < K; k++) {
                res += (float)matrix_a[m * K + k] * (float)matrix_b[k * N + n];
            }
            matrix_c[m * N + n] = (half)res;
        }
    }
}

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
#    define CUTE_ARCH_CP_ASYNC_SM80_ENABLED
#endif

__device__ __forceinline__ void LdMatrixX2(uint32_t *r, uint32_t addr) {
    asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(r[0]), "=r"(r[1]) : "r"(addr));
}

__device__ __forceinline__ void LdMatrixX4(uint32_t *r, uint32_t addr) {
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
                 : "=r"(r[0]), "=r"(r[1]), "=r"(r[2]), "=r"(r[3])
                 : "r"(addr));
}

__device__ __forceinline__ void MmaM16N8K16(uint32_t *c, uint32_t *a, uint32_t *b) {
#ifdef CUTE_ARCH_CP_ASYNC_SM80_ENABLED
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
        "{ %0, %1 },"
        "{ %2, %3, %4, %5 },"
        "{ %6, %7 },"
        "{ %8, %9 };"
        : "=r"(c[0]), "=r"(c[1])
        : "r"(a[0]), "r"(a[1]), "r"(a[2]), "r"(a[3]), "r"(b[0]), "r"(b[1]), "r"(c[0]), "r"(c[1]));
#endif
}

inline __device__ __host__ size_t div_ceil(size_t a, size_t b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16
#define WARP_SIZE 32

__global__ void mmaNaiveKernel(const half *__restrict__ matrix_a, const half *__restrict__ matrix_b,
                               half *__restrict__ matrix_c, size_t M, size_t N, size_t K) {
    const size_t K_tiles = div_ceil(K, MMA_K);

    const size_t warp_row = blockIdx.y * MMA_M;
    const size_t warp_col = blockIdx.x * MMA_N;

    if (warp_row >= M || warp_col >= N) {
        return;
    }

    __shared__ half A_smem[MMA_M][MMA_K];
    __shared__ half B_smem[MMA_N][MMA_K];
    __shared__ half C_smem[MMA_M][MMA_N];

    const size_t lane_id = threadIdx.x % WARP_SIZE;

    uint32_t RC[2] = {0, 0};

#pragma unroll
    for (size_t i = 0; i < K_tiles; ++i) {
        *((int4 *)(&A_smem[lane_id / 2][0]) + lane_id % 2) =
            *((int4 *)(&matrix_a[(warp_row + lane_id / 2) * K + i * MMA_K]) + lane_id % 2);

        if (lane_id < MMA_N * 2) {
            *((int4 *)(&B_smem[lane_id / 2][0]) + lane_id % 2) =
                *((int4 *)(&matrix_b[i * MMA_K + (warp_col + lane_id / 2) * K]) + lane_id % 2);
        }

        __syncthreads();

        uint32_t RA[4];
        uint32_t RB[2];

        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 8]);
        LdMatrixX4(RA, A_smem_lane_addr);

        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
        LdMatrixX2(RB, B_smem_lane_addr);

        MmaM16N8K16(RC, RA, RB);

        __syncthreads();
    }
    // store
    *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4)     = RC[0];
    *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

    __syncthreads();

    if (lane_id < MMA_M) {
        *((int4 *)(&matrix_c[(warp_row + lane_id) * N + warp_col])) = *((int4 *)(&C_smem[lane_id][0]));
    }
}

void MMAPTX(const half *matrix_a, const half *matrix_b, half *matrix_c, size_t M, size_t N, size_t K) {
    dim3 block(WARP_SIZE);
    dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));
    mmaNaiveKernel<<<grid, block>>>(matrix_a, matrix_b, matrix_c, M, N, K);
}

int main(int argc, char *argv[]) {
    int dev = 0;
    hipSetDevice(dev);

    const int M = 16;
    const int N = 8;
    const int K = 16;

    auto *matrix_a_host = new half[M * K]();
    auto *matrix_b_host = new half[K * N]();
    auto *matrix_c_host = new half[M * N]();

    GenerateRandomData(matrix_a_host, M, K);
    GenerateRandomData(matrix_b_host, K, N);
    // implement gemm with cpu
    Gemm(matrix_a_host, matrix_b_host, matrix_c_host, M, N, K);
    // convert matrix b from row-major to col-major, matrix_b[K, N] -> matrix[N, K]
    Transpose2D(matrix_b_host, K, N);
    half *matrix_a_device, *matrix_b_device, *matrix_c_device;
    CUDA_CHECK(hipMalloc(&matrix_a_device, M * K * sizeof(half)));
    CUDA_CHECK(hipMalloc(&matrix_b_device, K * N * sizeof(half)));
    CUDA_CHECK(hipMalloc(&matrix_c_device, M * N * sizeof(half)));
    CUDA_CHECK(hipMemcpy(matrix_a_device, matrix_a_host, M * K * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(matrix_b_device, matrix_b_host, K * N * sizeof(half), hipMemcpyHostToDevice));
    half alpha_device = 1.0f;
    half beta_device  = 0.0f;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    CUBLAS_CHECK(status);
    status = hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha_device, matrix_a_device, HIP_R_16F, K,
                          matrix_b_device, HIP_R_16F, K, &beta_device, matrix_c_device, HIP_R_16F, M, HIP_R_16F,
                          HIPBLAS_GEMM_DEFAULT);
    CUBLAS_CHECK(status);

    auto *matrix_c_host_cublas = new half[M * N]();
    CUDA_CHECK(hipMemcpy(matrix_c_host_cublas, matrix_c_device, M * N * sizeof(half), hipMemcpyDeviceToHost));
    // convert matrix c from col-major to row-major
    Transpose2D(matrix_c_host_cublas, N, M);
    printf("compare cpu with hipblasGemmEx\n");
    // CheckResult(matrix_c_host, matrix_c_host_cublas, M * N);

    // invoke mma ptx
    half *matrix_c_device_ptx = nullptr;
    CUDA_CHECK(hipMalloc(&matrix_c_device_ptx, M * N * sizeof(half)));
    MMAPTX(matrix_a_device, matrix_b_device, matrix_c_device_ptx, M, N, K);
    auto *matrix_c_host_ptx = new half[M * N]();
    CUDA_CHECK(hipMemcpy(matrix_c_host_ptx, matrix_c_device_ptx, M * N * sizeof(half), hipMemcpyDeviceToHost));
    printf("compare cpu with ptx mma\n");
    CheckResult(matrix_c_host, matrix_c_host_ptx, M * N);

    hipFree(matrix_a_device);
    hipFree(matrix_b_device);
    hipFree(matrix_c_device);
    hipFree(matrix_c_device_ptx);
    hipblasDestroy(handle);

    delete[] matrix_a_host;
    delete[] matrix_b_host;
    delete[] matrix_c_host;
    delete[] matrix_c_host_cublas;
    delete[] matrix_c_host_ptx;
}