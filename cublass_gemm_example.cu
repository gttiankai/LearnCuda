#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <random>

#include "cuda_utils.cuh"

void GenerateRandomData(float *matrix, int m, int n) {
    assert(matrix != nullptr);
    assert(m >= 0);
    assert(n >= 0);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            matrix[i * n + j] = dis(gen);
        }
    }
}

void Transpose2D(float *matrix, const int M, const int N) {
    assert(matrix != nullptr);
    auto buffer = new float[M * N]();
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            buffer[n * M + m] = matrix[m * N + n];
        }
    }
    memcpy(matrix, buffer, M * N * sizeof(float));
    delete[] buffer;
}

void Gemm(const float *matrix_a, const float *matrix_b, float *matrix_c, const int M, const int N, const int K) {
    assert(matrix_a != nullptr);
    assert(matrix_b != nullptr);
    assert(matrix_c != nullptr);
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float res = 0.0f;
            for (int k = 0; k < K; k++) {
                res += matrix_a[m * K + k] * matrix_b[k * N + n];
            }
            matrix_c[m * N + n] = res;
        }
    }
}

int main(int argc, char *argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    const int M = 64;
    const int N = 128;
    const int K = 256;

    auto *matrix_a_host = new float[M * K]();
    auto *matrix_b_host = new float[K * N]();
    auto *matrix_c_host = new float[M * N]();

    GenerateRandomData(matrix_a_host, M, K);
    GenerateRandomData(matrix_b_host, K, N);
    // implement gemm with cpu
    Gemm(matrix_a_host, matrix_b_host, matrix_c_host, M, N, K);

    Transpose2D(matrix_a_host, M, K);
    float *matrix_a_device, *matrix_b_device, *matrix_c_device;
    CUDA_CHECK(hipMalloc(&matrix_a_device, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc(&matrix_b_device, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&matrix_c_device, M * N * sizeof(float)));
    CUDA_CHECK(hipMemcpy(matrix_a_device, matrix_a_host, M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(matrix_b_device, matrix_b_host, K * N * sizeof(float), hipMemcpyHostToDevice));
    //    float *alpha_device;
    //    float *beta_device;
    //    CUDA_CHECK(hipMalloc(&alpha_device, N * sizeof(float)));
    //    CUDA_CHECK(hipMalloc(&beta_device, N * sizeof(float)));
    //    CUDA_CHECK(hipMemset(alpha_device, 1.0f, N * sizeof(float)));
    //    CUDA_CHECK(hipMemset(beta_device, 0.0f, N * sizeof(float)));
    float alpha_device = 1.0f;
    float beta_device  = 0.0f;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS error: " << status << " at " << __FILE__ << ":" << __LINE__ << std::endl;
        return -1;
    }
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, N, K, &alpha_device, matrix_a_device, HIP_R_32F, M,
                          matrix_b_device, HIP_R_32F, N, &beta_device, matrix_c_device, HIP_R_32F, M, HIP_R_32F,
                          HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        hipError_t error = hipGetLastError();
        std::cerr << "CUBLAS error: " << __FILE__ << ":" << __LINE__ << " line get cublas error(" << status << ") "
                  << hipGetErrorString(error) << std::endl;
        return -1;
    }

    auto *matrix_c_host_check = new float[M * N]();
    CUDA_CHECK(hipMemcpy(matrix_c_host_check, matrix_c_device, M * N * sizeof(float), hipMemcpyDeviceToHost));
    // checkResult(matrix_c_host, matrix_c_host_check, M * N);
    Transpose2D(matrix_c_host_check, N, M);
    checkResult(matrix_c_host, matrix_c_host_check, M * N);

    hipFree(matrix_a_device);
    hipFree(matrix_b_device);
    hipFree(matrix_c_device);
    hipblasDestroy(handle);

    delete[] matrix_a_host;
    delete[] matrix_b_host;
    delete[] matrix_c_host;
    delete[] matrix_c_host_check;
}