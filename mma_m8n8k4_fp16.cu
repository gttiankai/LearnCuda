// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <random>

#include "cuda_utils.cuh"

void GenerateRandomData(half *matrix, int m, int n) {
    assert(matrix != nullptr);
    assert(m >= 0);
    assert(n >= 0);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            matrix[i * n + j] = (half)dis(gen);
        }
    }
}

void Transpose2D(half *matrix, const int M, const int N) {
    assert(matrix != nullptr);
    auto buffer = new half[M * N]();
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            buffer[n * M + m] = matrix[m * N + n];
        }
    }
    memcpy(matrix, buffer, M * N * sizeof(half));
    delete[] buffer;
}

void Gemm(const half *matrix_a, const half *matrix_b, half *matrix_c, const int M, const int N, const int K) {
    assert(matrix_a != nullptr);
    assert(matrix_b != nullptr);
    assert(matrix_c != nullptr);
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float res = 0.0f;
            for (int k = 0; k < K; k++) {
                res += (float)matrix_a[m * K + k] * (float)matrix_b[k * N + n];
            }
            matrix_c[m * N + n] = (half)res;
        }
    }
}

int main(int argc, char *argv[]) {
    int dev = 0;
    hipSetDevice(dev);
    const int M = 64;
    const int N = 128;
    const int K = 256;

    auto *matrix_a_host = new half[M * K]();
    auto *matrix_b_host = new half[K * N]();
    auto *matrix_c_host = new half[M * N]();

    GenerateRandomData(matrix_a_host, M, K);
    GenerateRandomData(matrix_b_host, K, N);
    // implement gemm with cpu
    Gemm(matrix_a_host, matrix_b_host, matrix_c_host, M, N, K);

    Transpose2D(matrix_a_host, M, K);
    half *matrix_a_device, *matrix_b_device, *matrix_c_device;
    CUDA_CHECK(hipMalloc(&matrix_a_device, M * K * sizeof(half)));
    CUDA_CHECK(hipMalloc(&matrix_b_device, K * N * sizeof(half)));
    CUDA_CHECK(hipMalloc(&matrix_c_device, M * N * sizeof(half)));
    CUDA_CHECK(hipMemcpy(matrix_a_device, matrix_a_host, M * K * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(matrix_b_device, matrix_b_host, K * N * sizeof(half), hipMemcpyHostToDevice));
    //    half *alpha_device;
    //    half *beta_device;
    //    CUDA_CHECK(hipMalloc(&alpha_device, N * sizeof(half)));
    //    CUDA_CHECK(hipMalloc(&beta_device, N * sizeof(half)));
    //    CUDA_CHECK(hipMemset(alpha_device, 1.0f, N * sizeof(half)));
    //    CUDA_CHECK(hipMemset(beta_device, 0.0f, N * sizeof(half)));
    half alpha_device = 1.0f;
    half beta_device  = 0.0f;
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CUBLAS error: " << status << " at " << __FILE__ << ":" << __LINE__ << std::endl;
        return -1;
    }
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, M, N, K, &alpha_device, matrix_a_device, HIP_R_16F, M,
                          matrix_b_device, HIP_R_16F, N, &beta_device, matrix_c_device, HIP_R_16F, M, HIP_R_16F,
                          HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        hipError_t error = hipGetLastError();
        std::cerr << "CUBLAS error: " << __FILE__ << ":" << __LINE__ << " line get cublas error(" << status << ") "
                  << hipGetErrorString(error) << std::endl;
        return -1;
    }

    auto *matrix_c_host_check = new half[M * N]();
    CUDA_CHECK(hipMemcpy(matrix_c_host_check, matrix_c_device, M * N * sizeof(half), hipMemcpyDeviceToHost));
    // checkResult(matrix_c_host, matrix_c_host_check, M * N);
    Transpose2D(matrix_c_host_check, N, M);
    CheckResult(matrix_c_host, matrix_c_host_check, M * N);

    hipFree(matrix_a_device);
    hipFree(matrix_b_device);
    hipFree(matrix_c_device);
    hipblasDestroy(handle);

    delete[] matrix_a_host;
    delete[] matrix_b_host;
    delete[] matrix_c_host;
    delete[] matrix_c_host_check;
}